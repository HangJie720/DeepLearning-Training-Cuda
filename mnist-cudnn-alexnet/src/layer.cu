#include "hip/hip_runtime.h"
#include "layer.h"

#include <random>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cassert>
#include <math.h>
#include <algorithm>

#include <sstream>
#include <fstream>
#include <iostream>

using namespace cudl;

__global__ void add(float *array_a, float *array_b, float *array_c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (int i = tid; i < size; i += step) {
        array_c[i] = array_a[i] + array_b[i];
    }
}

/****************************************************************
 * Layer definition                                             *
 ****************************************************************/
Layer::Layer() {
    /* do nothing */
}

Layer::~Layer() {
#if (DEBUG_FORWARD > 0 || DEBUG_BACKWARD > 0)
    std::cout << "Destroy Layer: " << name_ << std::endl;
#endif

    if (output_ != nullptr) delete output_;
    if (grad_input_ != nullptr) delete grad_input_;

    if (weights_ != nullptr) delete weights_;
    if (biases_ != nullptr) delete biases_;
    if (grad_weights_ != nullptr) delete grad_weights_;
    if (grad_biases_ != nullptr) delete grad_biases_;
}

void Layer::init_weight_bias(unsigned int seed) {
    checkCudaErrors(hipDeviceSynchronize());

    if (weights_ == nullptr || biases_ == nullptr)
        return;

    // Create random network
    std::random_device rd;
    std::mt19937 gen(seed == 0 ? rd() : static_cast<unsigned int>(seed));

    // He uniform distribution
    float range = sqrt(6.f / input_->size());    // He's initialization
    std::uniform_real_distribution<> dis(-range, range);

    for (int i = 0; i < weights_->len(); i++)
        weights_->ptr()[i] = static_cast<float>(dis(gen));
    for (int i = 0; i < biases_->len(); i++)
        biases_->ptr()[i] = 0.f;

    // copy initialized value to the device
    weights_->to(DeviceType::cuda);
    biases_->to(DeviceType::cuda);

    std::cout << ".. initialized " << name_ << " layer .." << std::endl;
}

void Layer::update_weights_biases(float learning_rate) {
    float eps = -1.f * learning_rate;
    if (weights_ != nullptr && grad_weights_ != nullptr) {
#if (DEBUG_UPDATE)
        weights_->print(name_ + "::weights (before update)", true);
        grad_weights_->print(name_ + "::gweights", true);
#endif // DEBUG_UPDATE

        // w = w + eps * dw
        checkCublasErrors(
                hipblasSaxpy(cuda_->cublas(),
                            weights_->len(),
                            &eps,
                            grad_weights_->cuda(), 1,
                            weights_->cuda(), 1));

#if (DEBUG_UPDATE)
        weights_->print(name_ + "weights (after update)", true);
        // getchar();
#endif // DEBUG_UPDATE
    }

    if (biases_ != nullptr && grad_biases_ != nullptr) {
#if (DEBUG_UPDATE)
        biases_->print(name_ + "biases (before update)", true);
        grad_biases_->print(name_ + "gbiases", true);
#endif // DEBUG_UPDATE

        // b = b + eps * db
        checkCublasErrors(
                hipblasSaxpy(cuda_->cublas(),
                            biases_->len(),
                            &eps,
                            grad_biases_->cuda(), 1,
                            biases_->cuda(), 1));

#if (DEBUG_UPDATE)
        biases_->print(name_ + "biases (after update)", true);
        // getchar();
#endif // DEBUG_UPDATE
    }
}

float Layer::get_loss(Blob<float> *target) {
    assert("No Loss layer has no loss." && false);
    return EXIT_FAILURE;
}

int Layer::get_accuracy(Blob<float> *target) {
    assert("No Loss layer cannot estimate accuracy." && false);
    return EXIT_FAILURE;
}

int Layer::load_parameter() {
    std::stringstream filename_weights, filename_biases;

    // load weights and biases pretrained parameters
    filename_weights << name_ << ".bin";
    if (weights_->file_read(filename_weights.str()))
        return -1;

    filename_biases << name_ << ".bias.bin";
    if (biases_->file_read(filename_biases.str()))
        return -2;

    std::cout << ".. loaded " << name_ << " pretrain parameter.." << std::endl;

    return 0;
}

int Layer::save_parameter() {
    std::stringstream filename_weights, filename_biases;

    std::cout << ".. saving " << name_ << " parameter ..";

    // Write weights file
    if (weights_) {
        filename_weights << name_ << ".bin";
        if (weights_->file_write(filename_weights.str()))
            return -1;
    }

    // Write bias file
    if (biases_) {
        filename_biases << name_ << ".bias.bin";
        if (biases_->file_write(filename_biases.str()))
            return -2;
    }

    std::cout << " done .." << std::endl;

    return 0;
}

/****************************************************************
 * Dense Layer                                                  *
 ****************************************************************/

Dense::Dense(std::string name, int output_size) {
    name_ = name;
    output_size_ = output_size;
}

Dense::~Dense() {
    if (d_one_vec != nullptr)
        hipFree(d_one_vec);
}

__global__ void init_one_vec(float *d_one_vec, size_t length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= length) return;

    d_one_vec[i] = 1.f;
}

Blob<float> *Dense::forward(Blob<float> *input) {
    // initialize weights and biases
    if (weights_ == nullptr) {
        // setup parameter size information
        input_size_ = input->c() * input->h() * input->w();

        // initialize weight, bias, and output
        weights_ = new Blob<float>(1, 1, input_size_, output_size_);
        biases_ = new Blob<float>(1, 1, output_size_);

    }

    // initilaize input and output
    if (input_ == nullptr || batch_size_ != input->n()) {
        input_ = input;
        batch_size_ = input->n();

        if (output_ == nullptr)
            output_ = new Blob<float>(batch_size_, output_size_);
        else
            output_->reset(batch_size_, output_size_);

        output_->tensor();

        if (d_one_vec != nullptr)
            hipFree(d_one_vec);
        checkCudaErrors(hipMalloc((void **) &d_one_vec, sizeof(float) * batch_size_));
        init_one_vec << < (batch_size_ + BLOCK_DIM_1D - 1) / BLOCK_DIM_1D, BLOCK_DIM_1D >> > (d_one_vec, batch_size_);

        // initialize weights and biases
        if (load_pretrain_ && !freeze_) {
            if (load_parameter()) {
                std::cout << "error occurred.." << std::endl;
                exit(-1);
            }
        } else if (!freeze_) {
            init_weight_bias();
        } else {
            /* do nothing */
        }
    }

    // output = weights^T * input (without biases)
    checkCublasErrors(
            hipblasSgemm(cuda_->cublas(),
                        HIPBLAS_OP_T, HIPBLAS_OP_N,
                        output_size_, batch_size_, input_size_,
                        &cuda_->one,
                        weights_->cuda(), input_size_,
                        input_->cuda(), input_size_,
                        &cuda_->zero,
                        output_->cuda(), output_size_));

    // output += biases * d_one_vec^T
    checkCublasErrors(hipblasSgemm(cuda_->cublas(),
                                  HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  output_size_, batch_size_, 1,
                                  &cuda_->one,
                                  biases_->cuda(), output_size_,
                                  d_one_vec, 1,
                                  &cuda_->one,
                                  output_->cuda(), output_size_));


#if (DEBUG_DENSE & 0x01)
    input_->print(  name_ + "::input",  true);
    weights_->print(name_ + "::weight", true);
    biases_->print( name_ + "::bias",   true);
    output_->print( name_ + "::output", true);
#endif // DEBUG_DENSE

    return output_;
}

Blob<float> *Dense::backward(Blob<float> *grad_output) {
    if (grad_weights_ == nullptr) {
        grad_weights_ = new Blob<float>(weights_->shape());
        grad_biases_ = new Blob<float>(biases_->shape());
    }

    if (grad_input_ == nullptr || batch_size_ != grad_output->n()) {
        grad_output_ = grad_output;

        if (grad_input_ == nullptr)
            grad_input_ = new Blob<float>(input_->shape());
        else
            grad_input_->reset(input_->shape());
    }

    // db = (dy) * d_one_vec
    hipblasSgemv(cuda_->cublas(),
                HIPBLAS_OP_N,
                output_size_, batch_size_,
                &cuda_->one,
                grad_output_->cuda(), output_size_,
                d_one_vec, 1,
                &cuda_->zero,
                grad_biases_->cuda(), 1);

    // dw = x * (dy)^T
    hipblasSgemm(cuda_->cublas(),
                HIPBLAS_OP_N, HIPBLAS_OP_T,
                input_size_, output_size_, batch_size_,
                &cuda_->one,
                input_->cuda(), input_size_,
                grad_output_->cuda(), output_size_,
                &cuda_->zero,
                grad_weights_->cuda(), input_size_);

    // dx = W * dy
    if (!gradient_stop_)
        hipblasSgemm(cuda_->cublas(),
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    input_size_, batch_size_, output_size_,
                    &cuda_->one,
                    weights_->cuda(), input_size_,
                    grad_output_->cuda(), output_size_,
                    &cuda_->zero,
                    grad_input_->cuda(), input_size_);

#if (DEBUG_DENSE & 0x02)
    std::cout << name_ << "[BACKWARD]" << std::endl;
    grad_output->print(  name_ + "::gradients", true, grad_output->n());
    grad_weights_->print(name_ + "::gfilter", true);
    grad_biases_->print( name_ + "::gbias", true);
    if (!gradient_stop_)
        grad_input_->print(  name_ + "::gdata", true);
#endif // DEBUG_DENSE

    return grad_input_;
}

/****************************************************************
 * Activation Layer                                             *
 ****************************************************************/

Activation::Activation(std::string name, hipdnnActivationMode_t mode, float coef) {
    name_ = name;
    mode_ = mode;
    coef_ = coef;

    hipdnnCreateActivationDescriptor(&act_desc_);
    hipdnnSetActivationDescriptor(act_desc_, mode, HIPDNN_PROPAGATE_NAN, coef);
}

Activation::~Activation() {
    hipdnnDestroyActivationDescriptor(act_desc_);
}

Blob<float> *Activation::forward(Blob<float> *input) {
    if (input_ == nullptr || batch_size_ != input->n()) {
        input_ = input;
        input_desc_ = input->tensor();
        batch_size_ = input->n();

        if (output_ == nullptr)
            output_ = new Blob<float>(input->shape());
        else
            output_->reset(input->shape());

        output_desc_ = output_->tensor();
    }

    hipdnnActivationForward(cuda_->cudnn(),
                           act_desc_,
                           &cuda_->one,
                           input_desc_,
                           input->cuda(),
                           &cuda_->zero,
                           output_desc_,
                           output_->cuda());

    return output_;
}

Blob<float> *Activation::backward(Blob<float> *grad_output) {
    if (grad_input_ == nullptr || batch_size_ != grad_output->n()) {
        grad_output_ = grad_output;

        if (grad_input_ == nullptr)
            grad_input_ = new Blob<float>(input_->shape());
        else
            grad_input_->reset(input_->shape());
    }

    hipdnnActivationBackward(cuda_->cudnn(),
                            act_desc_,
                            &cuda_->one,
                            output_desc_, output_->cuda(),
                            output_desc_, grad_output->cuda(),
                            input_desc_, input_->cuda(),
                            &cuda_->zero,
                            input_desc_, grad_input_->cuda());

    return grad_input_;
}

/****************************************************************
 * Softmax definition                                           *
 ****************************************************************/

Softmax::Softmax(std::string name) {
    name_ = name;
}

Softmax::~Softmax() {

}

Blob<float> *Softmax::forward(Blob<float> *input) {
    if (input_ == nullptr || batch_size_ != input->n()) {
        input_ = input;
        input_desc_ = input->tensor();
        batch_size_ = input->n();

        if (output_ == nullptr)
            output_ = new Blob<float>(input->shape());
        else
            output_->reset(input->shape());

        output_desc_ = output_->tensor();
    }

#if (DEBUG_SOFTMAX & 0x01)
        std::cout << name_ << "[FORWARD]" << std::endl;
        input_->print(name_ + "::input", true, input->n());
#endif

    checkCudnnErrors(
            hipdnnSoftmaxForward(cuda_->cudnn(), HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
                                &cuda_->one, input_desc_, input->cuda(),
                                &cuda_->zero, output_desc_, output_->cuda()));

#if (DEBUG_SOFTMAX & 0x01)
    output_->print(name_ + "::output", true, input->n());
#endif

    return output_;
}

Blob<float> *Softmax::backward(Blob<float> *target) {
    checkCudaErrors(hipDeviceSynchronize());

    if (grad_input_ == nullptr || batch_size_ != target->n()) {
        if (grad_input_ == nullptr)
            grad_input_ = new Blob<float>(input_->shape());
        else
            grad_input_->reset(input_->shape());
    }

    // set grad_input_ as predict
    checkCudaErrors(hipMemcpyAsync(grad_input_->cuda(),
                                    output_->cuda(), output_->buf_size(),
                                    hipMemcpyDeviceToDevice));
    // set grad_input_ = predict - target
    checkCublasErrors(
            hipblasSaxpy(cuda_->cublas(), target->len(),
                        &cuda_->minus_one, target->cuda(), 1,
                        grad_input_->cuda(), 1));

    // normalize the grad_output by the batch size
    int grad_output_size = target->n() * target->c() * target->h() * target->w();
    float scale = 1.f / static_cast<float>(target->n());
    checkCublasErrors(hipblasSscal(cuda_->cublas(), grad_output_size, &scale, grad_input_->cuda(), 1));

#if (DEBUG_SOFTMAX & 0x02)
    std::cout << name_ << "[BACKWARD]" << std::endl;
    input_->print( name_ + "::input", true);
    output_->print(name_ + "::predict", true);
    target->print( name_ + "::y", true, target->n());
    grad_input_->print(name_ + "::dx", true, target->n());
#endif

    return grad_input_;
}

float Softmax::get_loss(Blob<float> *target) {
    return loss_.loss(output_, target);
}

int Softmax::get_accuracy(Blob<float> *target) {
    int batch_size = output_->n();
    int output_size = output_->size();

    assert(batch_size == target->n());
    assert(output_size == target->size());

    float *h_output, *h_target;
    int idx_output, idx_target;
    int hit_count = 0;

    // get predicts and targets
    h_output = output_->to(host);
    h_target = target->to(host);

    // idx_output = idx_target = 0;
    for (int b = 0; b < batch_size; b++) {
        idx_output = 0;
        idx_target = 0;

        for (int i = 1; i < 10; i++) {
            if (h_output[b * output_size + i] > h_output[b * output_size + idx_output])
                idx_output = i;
            if (h_target[b * output_size + i] > h_target[b * output_size + idx_target])
                idx_target = i;
        }

        if (idx_output == idx_target)
            hit_count++;
    }

    return hit_count;
}

/****************************************************************
 * Layer definition                                             *
 ****************************************************************/

/**
 * Convolutional layer with bias
 */
Conv2D::Conv2D(std::string name,
               int out_channels,
               int kernel_size,
               int stride,
               int padding,
               int dilation) :
        out_channels_(out_channels),
        kernel_size_(kernel_size),
        stride_(stride),
        padding_(padding),
        dilation_(dilation) {
    name_ = name;

    // create cudnn container handles
    hipdnnCreateFilterDescriptor(&filter_desc_);

    hipdnnCreateConvolutionDescriptor(&conv_desc_);
    checkCudnnErrors(hipdnnSetConvolution2dDescriptor(conv_desc_,
                                                     padding_, padding_, stride_, stride_, dilation_, dilation_,
                                                     HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
}

Conv2D::~Conv2D() {
    // distroy cudnn container resources
    hipdnnDestroyFilterDescriptor(filter_desc_);
    hipdnnDestroyConvolutionDescriptor(conv_desc_);

    // terminate internal created blobs
    if (d_workspace != nullptr) hipFree(d_workspace);
}

void Conv2D::set_workspace() {
    size_t temp_size = 0;

    // forward
    checkCudnnErrors(hipdnnGetConvolutionForwardAlgorithm(cuda_->cudnn(),
                                                         input_desc_, filter_desc_, conv_desc_, output_desc_,
                                                         HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &conv_fwd_algo_));
    checkCudnnErrors(hipdnnGetConvolutionForwardWorkspaceSize(cuda_->cudnn(),
                                                             input_desc_, filter_desc_, conv_desc_, output_desc_,
                                                             conv_fwd_algo_, &temp_size));
    workspace_size = std::max(workspace_size, temp_size);

    // todo trainable check
    // bwd - filter
    checkCudnnErrors(hipdnnGetConvolutionBackwardFilterAlgorithm(cuda_->cudnn(),
                                                                input_desc_, output_desc_, conv_desc_, filter_desc_,
                                                                HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0,
                                                                &conv_bwd_filter_algo_));
    checkCudnnErrors(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cuda_->cudnn(),
                                                                    input_desc_, output_desc_, conv_desc_, filter_desc_,
                                                                    conv_bwd_filter_algo_, &temp_size));
    workspace_size = std::max(workspace_size, temp_size);

    // bwd - data
    checkCudnnErrors(hipdnnGetConvolutionBackwardDataAlgorithm(cuda_->cudnn(),
                                                              filter_desc_, output_desc_, conv_desc_, input_desc_,
                                                              HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0,
                                                              &conv_bwd_data_algo_));
    checkCudnnErrors(hipdnnGetConvolutionBackwardDataWorkspaceSize(cuda_->cudnn(),
                                                                  filter_desc_, output_desc_, conv_desc_, input_desc_,
                                                                  conv_bwd_data_algo_, &temp_size));
    workspace_size = std::max(workspace_size, temp_size);

    if (workspace_size > 0) {
        if (d_workspace != nullptr) checkCudaErrors(hipFree(d_workspace));
        checkCudaErrors(hipMalloc((void **) &d_workspace, workspace_size));
    }
}

Blob<float> *Conv2D::forward(Blob<float> *input) {
    // initialize weights and bias
    if (weights_ == nullptr) {
        // initialize containers handles
        checkCudnnErrors(hipdnnSetFilter4dDescriptor(filter_desc_,
                                                    HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                                    out_channels_, input->c(), kernel_size_, kernel_size_));

        weights_ = new Blob<float>(out_channels_, input->c(), kernel_size_, kernel_size_);
        biases_ = new Blob<float>(1, out_channels_);    // bias size
        bias_desc_ = biases_->tensor();
    }

    // initilaize input and output
    if (input_ == nullptr || batch_size_ != input->n()) {
        // initialize input
        input_ = input;
        input_desc_ = input->tensor();
        batch_size_ = input->n();

        // initilaize output
        checkCudnnErrors(hipdnnGetConvolution2dForwardOutputDim(
                conv_desc_, input_desc_, filter_desc_,
                &output_size_[0], &output_size_[1], &output_size_[2], &output_size_[3]));
        if (output_ == nullptr)
            output_ = new Blob<float>(output_size_);
        else
            output_->reset(output_size_);

        output_desc_ = output_->tensor();

        // initialize workspace for cudnn
        set_workspace();

        // initialize weights
        if (load_pretrain_ && !freeze_) {
            if (load_parameter()) {
                std::cout << "error occurred.." << std::endl;
                exit(-1);
            }
        } else if (!freeze_) {
            init_weight_bias();
        } else {
            /* do nothing */
        }
    }

    checkCudnnErrors(hipdnnConvolutionForward(cuda_->cudnn(),
                                             &cuda_->one, input_desc_, input_->cuda(),
                                             filter_desc_, weights_->cuda(), conv_desc_, conv_fwd_algo_, d_workspace,
                                             workspace_size,
                                             &cuda_->zero, output_desc_, output_->cuda()));

    checkCudnnErrors(hipdnnAddTensor(cuda_->cudnn(),
                                    &cuda_->one, bias_desc_, biases_->cuda(),
                                    &cuda_->one, output_desc_, output_->cuda()));

#if (DEBUG_CONV & 0x01)
    input_->print(  name_ + "::input", true, input_->n(), 28);
    weights_->print(name_ + "::weight", true);
    biases_->print( name_ + "::bias", true);
    output_->print( name_ + "::output", true);
#endif

    return output_;
}

Blob<float> *Conv2D::backward(Blob<float> *grad_output) {
    // initialize grad_output back-propagation space
    if (grad_input_ == nullptr || batch_size_ != grad_output->n()) {
        grad_output_ = grad_output;
        grad_weights_ = new Blob<float>(weights_->shape());
        grad_biases_ = new Blob<float>(1, biases_->c());

        if (grad_input_ == nullptr)
            grad_input_ = new Blob<float>(input_->shape());
        else
            grad_input_->reset(input_->shape());
    }

    // gradients of biases
    checkCudnnErrors(
            hipdnnConvolutionBackwardBias(cuda_->cudnn(),
                                         &cuda_->one,
                                         output_desc_, grad_output->cuda(),
                                         &cuda_->zero,
                                         bias_desc_, grad_biases_->cuda()));

    // gradients of weights
    checkCudnnErrors(
            hipdnnConvolutionBackwardFilter(cuda_->cudnn(),
                                           &cuda_->one,
                                           input_desc_, input_->cuda(),
                                           output_desc_, grad_output_->cuda(),
                                           conv_desc_, conv_bwd_filter_algo_, d_workspace, workspace_size,
                                           &cuda_->zero,
                                           filter_desc_, grad_weights_->cuda()));

    // gradients of input data
    if (!gradient_stop_) checkCudnnErrors(
            hipdnnConvolutionBackwardData(cuda_->cudnn(),
                                         &cuda_->one,
                                         filter_desc_, weights_->cuda(),
                                         output_desc_, grad_output->cuda(),
                                         conv_desc_, conv_bwd_data_algo_, d_workspace, workspace_size,
                                         &cuda_->zero,
                                         input_desc_, grad_input_->cuda()));

#if (DEBUG_CONV & 0x02)
    std::cout << name_ << "[BACKWARD]" << std::endl;
    grad_output->print( name_ + "::gradients", true);
    grad_biases_->print(name_ + "gbias", true);
    grad_weights_->print(name_+ "gfilter", true);
    if (!gradient_stop_)
        grad_input_->print(name_+"gdata", true);
#endif

#if (DEBUG_CONV & 0x04)
    grad_output->print( name_ + "::gradients", true);
    grad_biases_->print( name_ + "::gbias", true);
#endif

    return grad_input_;
}

/****************************************************************
 * Layer definition                                             *
 ****************************************************************/

Pooling::Pooling(std::string name,
                 int kernel_size,
                 int padding,
                 int stride,
                 hipdnnPoolingMode_t mode) :
        kernel_size_(kernel_size),
        padding_(padding),
        stride_(stride),
        mode_(mode) {
    name_ = name;

    hipdnnCreatePoolingDescriptor(&pool_desc_);
    hipdnnSetPooling2dDescriptor(pool_desc_, mode_, HIPDNN_PROPAGATE_NAN,
                                kernel_size_, kernel_size_, padding_, padding_, stride_, stride_);
}

Pooling::~Pooling() {
    hipdnnDestroyPoolingDescriptor(pool_desc_);
}

Blob<float> *Pooling::forward(Blob<float> *input) {
    if (input_ == nullptr || batch_size_ != input->n()) {
        input_ = input;

        // resource initialize
        input_desc_ = input_->tensor();
        batch_size_ = input->n();

        // setting output
        hipdnnGetPooling2dForwardOutputDim(pool_desc_, input_desc_,
                                          &output_size_[0], &output_size_[1], &output_size_[2], &output_size_[3]);
        if (output_ == nullptr)
            output_ = new Blob<float>(output_size_);
        else
            output_->reset(output_size_);

        output_desc_ = output_->tensor();
    }

    hipdnnPoolingForward(cuda_->cudnn(), pool_desc_,
                        &cuda_->one, input_desc_, input_->cuda(),
                        &cuda_->zero, output_desc_, output_->cuda());

    return output_;
}

Blob<float> *Pooling::backward(Blob<float> *grad_output) {
    if (grad_input_ == nullptr || batch_size_ != grad_output->n()) {
        grad_output_ = grad_output;

        if (grad_input_ == nullptr)
            grad_input_ = new Blob<float>(input_->shape());
        else
            grad_input_->reset(input_->shape());
    }

    checkCudnnErrors(
            hipdnnPoolingBackward(cuda_->cudnn(), pool_desc_,
                                 &cuda_->one,
                                 output_desc_, output_->cuda(),
                                 output_desc_, grad_output->cuda(),
                                 input_desc_, input_->cuda(),
                                 &cuda_->zero,
                                 input_desc_, grad_input_->cuda()));

    return grad_input_;
}


/****************************************************************
 * Local Response Normalization definition                      *
 ****************************************************************/

LRN::LRN(std::string name, unsigned lrnN, double lrnAlpha, double lrnBeta, double lrnK) {
    name_ = name;
    lrnN_ = lrnN;
    lrnAlpha_ = lrnAlpha;
    lrnBeta_ = lrnBeta;
    lrnK_ = lrnK;

    checkCudnnErrors(hipdnnCreateLRNDescriptor(&normDesc_));
    checkCudnnErrors(hipdnnSetLRNDescriptor(normDesc_,
                                           lrnN_,
                                           lrnAlpha_,
                                           lrnBeta_,
                                           lrnK_));
}

LRN::~LRN() {
    hipdnnDestroyLRNDescriptor(normDesc_);
}

Blob<float> *LRN::forward(Blob<float> *input) {
    // initilaize input and output
    if (input_ == nullptr || batch_size_ != input->n()) {
        input_ = input;
        batch_size_ = input->n();
        input_desc_ = input_->tensor();

        if (output_ == nullptr)
            output_ = new Blob<float>(input->shape());
        else
            output_->reset(input->shape());
        output_desc_ = output_->tensor();
    }

    checkCudnnErrors(hipdnnLRNCrossChannelForward(cuda_->cudnn(),
                                                 normDesc_,
                                                 HIPDNN_LRN_CROSS_CHANNEL,
                                                 &cuda_->one,
                                                 input_desc_,
                                                 input_->cuda(),
                                                 &cuda_->zero,
                                                 output_desc_,
                                                 output_->cuda()));

#if (DEBUG_CONV & 0x01)
    input_->print(  name_ + "::input", true, input_->n(), 28);
    output_->print( name_ + "::output", true);
#endif

    return output_;
}

Blob<float> *LRN::backward(Blob<float> *grad_input) {
    // initialize grad_output back-propagation space
    if (grad_input_ == nullptr || batch_size_ != grad_input->n()) {
        grad_output_ = grad_input;

        if (grad_input_ == nullptr)
            grad_input_ = new Blob<float>(input_->shape());
        else
            grad_input_->reset(input_->shape());
    }


    checkCudnnErrors(hipdnnLRNCrossChannelBackward(cuda_->cudnn(),
                                                  normDesc_,
                                                  HIPDNN_LRN_CROSS_CHANNEL,
                                                  &cuda_->one,
                                                  output_desc_,
                                                  output_->cuda(),
                                                  output_desc_,
                                                  grad_output_->cuda(),
                                                  input_desc_,
                                                  input_->cuda(),
                                                  &cuda_->zero,
                                                  input_desc_,
                                                  grad_input_->cuda()));

    return grad_input_;
}


/****************************************************************
 * Dropout definition                      *
 ****************************************************************/

Dropout::Dropout(std::string name, float dropout) {
    name_ = name;
    dropout_ = dropout;

    checkCudnnErrors(hipdnnCreateDropoutDescriptor(&dropoutDesc_));

}

Dropout::~Dropout() {
    hipdnnDestroyDropoutDescriptor(dropoutDesc_);
    if (states_ != nullptr) hipFree(states_);
    if (m_pReserve_ != nullptr) hipFree(m_pReserve_);
}

Blob<float> *Dropout::forward(Blob<float> *input) {
    // initilaize input and output
    if (input_ == nullptr || batch_size_ != input->n()) {
        input_ = input;
        batch_size_ = input->n();
        input_desc_ = input_->tensor();

        if (output_ == nullptr)
            output_ = new Blob<float>(input->shape());
        else
            output_->reset(input->shape());
        output_desc_ = output_->tensor();

        checkCudnnErrors(hipdnnDropoutGetStatesSize(cuda_->cudnn(), &stateSize_));
        if (states_ != nullptr) checkCudaErrors(hipFree(states_));
        checkCudaErrors(hipMalloc((void **) &states_, stateSize_));
        checkCudnnErrors(cudnnDropoutGetReserveSpaceSize(input_desc_, &reserveSize_));
        if (m_pReserve_ != nullptr) checkCudaErrors(hipFree(m_pReserve_));
        checkCudaErrors(hipMalloc((void **) &m_pReserve_, reserveSize_));


        checkCudnnErrors(hipdnnSetDropoutDescriptor(dropoutDesc_,
                                                   cuda_->cudnn(),
                                                   dropout_,
                                                   states_,
                                                   stateSize_,
                                                   seed_));
    }

    checkCudnnErrors(cudnnDropoutForward(cuda_->cudnn(),
                                         dropoutDesc_,
                                         input_desc_,
                                         input_->cuda(),
                                         output_desc_,
                                         output_->cuda(),
                                         m_pReserve_,
                                         reserveSize_));
    return output_;
}

Blob<float> *Dropout::backward(Blob<float> *grad_input) {
    // initialize grad_output back-propagation space
    if (grad_input_ == nullptr || batch_size_ != grad_input->n()) {
        grad_output_ = grad_input;

        if (grad_input_ == nullptr)
            grad_input_ = new Blob<float>(input_->shape());
        else
            grad_input_->reset(input_->shape());
    }
    checkCudnnErrors(cudnnDropoutBackward(cuda_->cudnn(), dropoutDesc_,
                                          output_desc_, grad_output_->cuda(),
                                          input_desc_, grad_input_->cuda(),
                                          m_pReserve_, reserveSize_));

    return grad_input_;
}