#include "hip/hip_runtime.h"
#include "loss.h"
#include "helper.h"

#include <cassert>
#include <hip/hip_runtime.h>

using namespace cudl;

/* 
 * https://deepnotes.io/softmax-crossentropy
 * */

CrossEntropyLoss::CrossEntropyLoss()
{
    hipMalloc((void**)&d_loss_, sizeof(float));
}

CrossEntropyLoss::~CrossEntropyLoss()
{
    if (d_loss_ != nullptr)
        hipFree(d_loss_);
        d_loss_ = nullptr;

    if (d_workspace_ != nullptr)
        hipFree(d_workspace_);
}

__device__ float clip(float prediction, float epsilon=1e-12)
{
    return fmin(fmax(prediction, epsilon), 1.f - epsilon);
}

__global__ void
softmax_loss_kernel(float *reduced_loss, float *predict, float *target, float *workspace, int batch_size, int num_outputs)
{
    int batch_idx = blockDim.x * blockIdx.x + threadIdx.x;

    extern __shared__ float s_data[];
    float loss = 0.f;

    // each thread calculate entropy for each data and accumulate to shared memory
    if (batch_idx > 0)
        return;

    for (int c = 0; c < num_outputs; c++)
        loss += target[batch_idx * num_outputs + c] * logf(predict[batch_idx * num_outputs + c]);
    workspace[batch_idx] = -loss;

    // then, we do reduction the result to calculate loss using 1 thread block
    if (blockIdx.x > 0) return;

    // cumulate workspace data
    s_data[threadIdx.x] = 0.f;
    for (int i = 0; i < batch_size; i += blockDim.x)
    {
        s_data[threadIdx.x] += workspace[threadIdx.x + i];
    }

    __syncthreads();

    // reduction
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x + stride < batch_size)
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];

        __syncthreads();
    }

    if (threadIdx.x == 0) {
        reduced_loss[blockIdx.x] = s_data[0];
    }
}

void CrossEntropyLoss::init_workspace(int batch_size)
{
    if (d_workspace_ == nullptr)
        hipMalloc((void**)&d_workspace_, sizeof(float) * batch_size);
}

float CrossEntropyLoss::loss(Blob<float> *predict, Blob<float> *target)
{
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, softmax_loss_kernel, BLOCK_DIM_1D, BLOCK_DIM_1D * sizeof(float));

    int batch_size = target->n();
    int num_outputs = target->c();

    init_workspace(batch_size);

    #if (DEBUG_LOSS)
    std::cout << "[[ LOSS ]]" << std::endl;
    predict->print("predict", true);
    target->print("target", true);
    #endif // DEBUG_LOSS

    int num_blocks = min(num_blocks_per_sm * num_sms, \
                         (target->size() + BLOCK_DIM_1D - 1) / BLOCK_DIM_1D);
    softmax_loss_kernel<<< num_blocks, BLOCK_DIM_1D, BLOCK_DIM_1D * sizeof(float), 0 >>>
                (d_loss_, predict->cuda(), target->cuda(), d_workspace_, batch_size, num_outputs);
    hipMemcpy(&h_loss_, d_loss_, sizeof(float), hipMemcpyDeviceToHost);
    
    return h_loss_;
}


