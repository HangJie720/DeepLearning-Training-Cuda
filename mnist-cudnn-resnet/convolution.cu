#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <iostream>
#include "src/helper.h"

int main()
{
    hipdnnHandle_t cudnn;
    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t output_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnTensorDescriptor_t bias_desc;

    hipdnnConvolutionFwdAlgo_t falgo;
    hipdnnConvolutionBwdFilterAlgo_t b_falgo;
    hipdnnConvolutionBwdDataAlgo_t b_dalgo;

    float *d_input = nullptr;
    float *d_output = nullptr;
    float *d_filter = nullptr;
    float *d_bias = nullptr;

    int input_n = 64;
    int input_c = 1;
    int input_h = 28;
    int input_w = 28;

    // output size
    int output_n = input_n;
    int output_c = 20;
    int output_h = 1;
    int output_w = 1;

    // kernel size
    int filter_h = 5;
    int filter_w = 5;

    // alpha, beta
    float one = 1.f;
    float zero = 0.f;

    std::cout << "[" <<  __LINE__ << "]" << std::endl;

    hipdnnCreate(&cudnn);

    std::cout << "[" <<  __LINE__ << "]" << std::endl;

    /* Create Resources */
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnCreateTensorDescriptor(&output_desc);
    hipdnnCreateFilterDescriptor(&filter_desc);
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnCreateTensorDescriptor(&bias_desc);

    std::cout << "[" <<  __LINE__ << "]" << std::endl;

    // Initilziae resources
    hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_n, input_c, input_h, input_w);
    hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, output_c, input_c, filter_h, filter_w);
    hipdnnSetConvolution2dDescriptor(conv_desc,
                                    0, 0,
                                    1, 1,
                                    1, 1,
                                    HIPDNN_CROSS_CORRELATION,
                                    HIPDNN_DATA_FLOAT);
    hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, filter_desc, &output_n, &output_c, &output_h, &output_w);
    hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, output_n, output_c, output_h, output_w);
    hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, output_c, 1, 1);

    int weight_size = output_c * input_c * filter_h * filter_w;
    int bias_size = output_c;

    std::cout << "input  size: " << input_n << " " << input_c << " " << input_h << " " << input_w << std::endl;
    std::cout << "output size: " << output_n << " " << output_c << " " << output_h << " " << output_w << std::endl;

    std::cout << "[" <<  __LINE__ << "]" << std::endl;

    // convolution
    size_t workspace_size = 0;
    size_t temp_size = 0;
    float *d_workspace = nullptr;
    hipdnnGetConvolutionForwardAlgorithm(cudnn, input_desc, filter_desc, conv_desc, output_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &falgo);
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_desc, filter_desc, conv_desc, output_desc, falgo, &temp_size);
    workspace_size = max(workspace_size, temp_size);

    // convolution (bwd - filter)
    hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn, input_desc, output_desc, conv_desc, filter_desc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &b_falgo);
    hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, input_desc, output_desc, conv_desc, filter_desc, b_falgo, &temp_size);
    workspace_size = max(workspace_size, temp_size);

    // convolution (bwd - data)
    hipdnnGetConvolutionBackwardDataAlgorithm(cudnn, filter_desc, output_desc, conv_desc, input_desc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &b_dalgo);
    hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, filter_desc, output_desc, conv_desc, input_desc, b_dalgo, &temp_size);
    workspace_size = max(workspace_size, temp_size);

    std::cout << "workspace size: " << workspace_size << std::endl;
    std::cout << "[" <<  __LINE__ << "]" << std::endl;

    // allocate memory space
    hipMalloc((void**)&d_input,        sizeof(float) * input_n * input_c * input_h * input_w);
    hipMalloc((void**)&d_filter,       sizeof(float) * weight_size);
    hipMalloc((void**)&d_output,       sizeof(float) * output_n * output_c * output_h * output_w);
    hipMalloc((void**)&d_workspace,    sizeof(float) * workspace_size);
    hipMalloc((void**)&d_bias,         sizeof(float) * bias_size);

    std::cout << "[" <<  __LINE__ << "]" << std::endl;

    // Forward
    checkCudnnErrors(hipdnnConvolutionForward(cudnn, &one, input_desc, d_input, filter_desc, d_filter, conv_desc, falgo, d_workspace, workspace_size, &zero, output_desc, d_output));
    checkCudnnErrors(hipdnnAddTensor(cudnn, &one, bias_desc, d_bias, &one, output_desc, d_output));
    checkCudaErrors(hipGetLastError());
    
    std::cout << "[" <<  __LINE__ << "]" << std::endl;

    // backward
    checkCudnnErrors(hipdnnConvolutionBackwardBias(cudnn, &one, output_desc, d_output, &zero, bias_desc, d_bias));
    checkCudnnErrors(hipdnnConvolutionBackwardFilter(cudnn, &one, input_desc, d_input, output_desc, d_output, conv_desc, b_falgo, d_workspace, workspace_size, &zero, filter_desc, d_filter));
    checkCudnnErrors(hipdnnConvolutionBackwardData(cudnn, &one, filter_desc, d_filter, output_desc, d_output, conv_desc, b_dalgo, d_workspace, workspace_size, &zero, input_desc, d_input));
    checkCudaErrors(hipGetLastError());
    
    std::cout << "[" <<  __LINE__ << "]" << std::endl;

    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroyTensorDescriptor(bias_desc);

    std::cout << "[" <<  __LINE__ << "]" << std::endl;

    hipFree(d_input);    
    hipFree(d_filter);
    hipFree(d_output);
    hipFree(d_workspace);
    hipFree(d_bias);

    hipdnnDestroy(cudnn);

    std::cout << "[" <<  __LINE__ << "]" << std::endl;
}